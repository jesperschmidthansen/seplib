#include "hip/hip_runtime.h"

#include "sepcudamem.h"


sepcupart* sep_cuda_allocate_memory(unsigned npartPadding){
	sepcupart* ptr;
	
	if ( hipHostMalloc((void **)&ptr, sizeof(sepcupart))== hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	size_t nbytes = npartPadding*sizeof(float4);
	size_t nbytes_excludelist = (1+SEP_MAX_NUMB_EXCLUSION)*npartPadding*sizeof(int);

	// Host
	if ( hipHostMalloc((void **)&(ptr->hx), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hv), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hf), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hx0), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->ht), npartPadding*sizeof(char)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hexclusion), nbytes_excludelist) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hcrossings), npartPadding*sizeof(int3)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hmolindex), npartPadding*sizeof(int)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	// Device
	if ( hipMalloc((void **)&(ptr->dx), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->dv), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->df), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->dx0), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->ddist), npartPadding*sizeof(float)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->epot), npartPadding*sizeof(float)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->press), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();

	if ( hipMalloc((void **)&(ptr->sumpress), sizeof(float4)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->dexclusion), nbytes_excludelist) == hipErrorOutOfMemory )
		sep_cuda_mem_error();

	ptr->maxneighb = SEP_CUDA_MAXNEIGHBS;
	if ( hipMalloc(&(ptr->neighblist), sizeof(int)*npartPadding*(ptr->maxneighb)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();

	if ( hipMalloc((void **)&(ptr->dcrossings), npartPadding*sizeof(int3)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->dmolindex), npartPadding*sizeof(int)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	return ptr;
}

void sep_cuda_free_memory(sepcupart *ptr, sepcusys *sptr){
	
	// Particle structure
	hipHostFree(ptr->hx); 	hipHostFree(ptr->hv); 
	hipHostFree(ptr->hf); 	hipHostFree(ptr->hx0);
	hipHostFree(ptr->ht);
	
	hipHostFree(ptr->hexclusion); hipHostFree(ptr->hcrossings); hipHostFree(ptr->hmolindex); 
	
	hipFree(ptr->dx); hipFree(ptr->dv); hipFree(ptr->df); hipFree(ptr->dx0);
	hipFree(ptr->ddist); hipFree(ptr->neighblist);
	hipFree(ptr->epot); hipFree(ptr->press); hipFree(ptr->sumpress); 
	
	hipFree(ptr->dexclusion); hipFree(ptr->dcrossings); hipFree(ptr->dmolindex);
	
	
	hipHostFree(ptr);
	
	// System structure
	hipFree(sptr->denergies); hipHostFree(sptr->henergies);
	hipFree(sptr->dalpha); hipFree(sptr->dupdate); 
	
	hipHostFree(sptr);
}


sepcupart* sep_cuda_load_xyz(const char *xyzfile){
	unsigned npart;
	int nthreads = SEP_CUDA_NTHREADS;
	
	FILE *fin = fopen(xyzfile, "r");
	if ( fin == NULL )
		sep_cuda_file_error();
	
	fscanf(fin, "%d\n", &npart);
	
	unsigned nblocks = (npart + nthreads - 1) / nthreads;
	unsigned npartwithPadding = nblocks*nthreads;
	
	sepcupart *ptr = sep_cuda_allocate_memory(npartwithPadding);
		
	ptr->nblocks = nblocks; 
	ptr->nthreads = nthreads;
	ptr->npart = npart; 
	ptr->npart_padding = npartwithPadding;
	ptr->hexclusion_rule = SEP_CUDA_EXCL_NONE;
	
	fscanf(fin, "%f %f %f\n", &(ptr->lbox.x), &(ptr->lbox.y), &(ptr->lbox.z));
	
	for ( unsigned n=0; n<npart; n++ ) {
		fscanf(fin, "%c %f %f %f %f %f %f %f %f\n", 
			   &(ptr->ht[n]), &(ptr->hx[n].x),&(ptr->hx[n].y),&(ptr->hx[n].z), 
			   &(ptr->hv[n].x),&(ptr->hv[n].y),&(ptr->hv[n].z), &(ptr->hx[n].w), &(ptr->hv[n].w));
		ptr->hcrossings[n].x = ptr->hcrossings[n].y = ptr->hcrossings[n].z = 0;
		ptr->hmolindex[n] = -1;
	}
	
	fclose(fin);

	for ( unsigned n=npart; n<npartwithPadding; n++ ){
		ptr->hx[n].x = ptr->hx[n].y = ptr->hx[n].z = 0.0f;
		ptr->hv[n].x = ptr->hv[n].y = ptr->hv[n].z = 0.0f;
		ptr->hv[n].w = 1.0; ptr->ht[n] = 'A';
		ptr->hmolindex[n] = -1;
	}
	
	sep_cuda_copy(ptr, 'x', 'd'); 
	sep_cuda_copy(ptr, 'v', 'd');
	sep_cuda_copy(ptr, 'f', 'd');
	sep_cuda_copy(ptr, 'c', 'd');
	
	hipMemcpy(ptr->dmolindex, ptr->hmolindex, npartwithPadding*sizeof(int), hipMemcpyHostToDevice);

	return ptr;
}


sepcusys *sep_cuda_sys_setup(sepcupart *pptr){
	
	sepcusys *sptr;
	if ( hipHostMalloc((void **)&sptr, sizeof(sepcusys)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	sptr->npart = pptr->npart;
	sptr->npart_padding = pptr->npart_padding;
	sptr->nblocks = pptr->nblocks;
	sptr->nthreads = pptr->nthreads;
	sptr->dt = 0.005;
	sptr->skin = 0.3;
	sptr->lbox = pptr->lbox;
	
	if ( hipHostMalloc((void **)&(sptr->henergies), sizeof(float3)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(sptr->denergies), sizeof(float3)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(sptr->dalpha), sizeof(float)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	sep_cuda_setvalue<<<1,1>>>(sptr->dalpha, 0.2);
	
	
	if ( hipMalloc((void **)&(sptr->dupdate), sizeof(int)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	sep_cuda_setvalue<<<1,1>>>(sptr->dupdate, 1);
	
	return sptr;
}


