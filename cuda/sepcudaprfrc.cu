#include "hip/hip_runtime.h"

#include "sepcudaprfrc.h"


bool sep_cuda_check_neighblist(sepcupart *ptr, float maxdist){
		
	sep_cuda_sumdistance<<<ptr->nblocks,ptr->nthreads>>>(&(ptr->dsumdist), ptr->ddist, maxdist, ptr->npart);
	hipDeviceSynchronize();
	
	float sumdr=0.0f;
	hipMemcpy(&sumdr, &(ptr->dsumdist), sizeof(float), hipMemcpyDeviceToHost);
	
	float avsumdr = sumdr/ptr->npart;
		
	if ( avsumdr > maxdist ){
		sep_cuda_setvalue<<<1,1>>>(&(ptr->dsumdist), 0);
		hipDeviceSynchronize();
		return true;
	}	
	else 
		return false;
	
}

void sep_cuda_reset_exclusion(sepcupart *pptr){
	
	for ( unsigned n=0; n<pptr->npart_padding; n++ ){
		int offset = n*(SEP_MAX_NUMB_EXCLUSION+1);

		pptr->hexclusion[offset] = 0;
		for ( int m=1; m<=SEP_MAX_NUMB_EXCLUSION; m++ )
			pptr->hexclusion[offset+m] = -1;
		
	}
	
	sep_cuda_copy_exclusion(pptr);	
}

void sep_cuda_copy_exclusion(sepcupart *pptr){

	size_t nbytes_excludelist = (SEP_MAX_NUMB_EXCLUSION+1)*pptr->npart_padding*sizeof(int);
		
	hipError_t __err = hipMemcpy(pptr->dexclusion, pptr->hexclusion, nbytes_excludelist, hipMemcpyHostToDevice);
	if ( __err != hipSuccess ) fprintf(stderr, "Error copying\n");	

}

void sep_cuda_set_hexclusion(sepcupart *pptr, int a, int b){
	
	int offset_a = a*(SEP_MAX_NUMB_EXCLUSION+1); 
	int offset_lst = pptr->hexclusion[offset_a];
	
	pptr->hexclusion[offset_a + offset_lst + 1] = b;
	pptr->hexclusion[offset_a] = pptr->hexclusion[offset_a] + 1;
	
}



void sep_cuda_set_exclusion(sepcupart *aptr, const char rule[]){
	
	if ( strcmp(rule, "bonds")==0 ){
		aptr->hexclusion_rule = SEP_CUDA_EXCL_BONDS;
	}
	else if (strcmp(rule, "molecule")==0 ){
		aptr->hexclusion_rule = SEP_CUDA_EXCL_MOLECULE;
	}
	else {
		fprintf(stderr, "Not valid exclusion rule\n");
	}
	
	size_t nbytes = sizeof(unsigned);
	hipMemcpy(&(aptr->dexclusion_rule), &(aptr->hexclusion_rule), nbytes, hipMemcpyHostToDevice);
	
}


// Kernels

/* Neighbourlist for particles - no exclusion */
__global__ void sep_cuda_build_neighblist(int *neighlist, float4 *p, float *dist, float cf, 
										  float3 lbox, unsigned nneighmax, unsigned npart) {

	int pidx = blockDim.x * blockIdx.x + threadIdx.x;
		
	if ( pidx < npart ){
		float cfsqr = cf*cf; 
		int arrayOffset = pidx*nneighmax;
	
		float mpx = __ldg(&p[pidx].x); float mpy = __ldg(&p[pidx].y); float mpz = __ldg(&p[pidx].z);

		#pragma unroll	
		for ( int n=0; n<nneighmax; n++ ) neighlist[arrayOffset + n] = -1; //<- this should be optimized 
		
		dist[pidx] = 0.0f;
		
		int shift = 0;
		for ( int tile = 0; tile < gridDim.x; tile++ ) {

			/*
			__shared__ float4 spos[SEP_CUDA_NTHREADS];
			spos[threadIdx.x] = p[tile * blockDim.x + threadIdx.x];
			__syncthreads();
			*/
			
			for ( int j = 0; j < SEP_CUDA_NTHREADS; j++ ) {
				int idxj = tile*blockDim.x + j;
				
				if ( idxj >= npart )  break;

				/*
				float dx = mpx - spos[j].x; dx = sep_cuda_wrap(dx, lbox.x);
				float dy = mpy - spos[j].y; dy = sep_cuda_wrap(dy, lbox.y);
				float dz = mpz - spos[j].z; dz = sep_cuda_wrap(dz, lbox.z);
				*/
				
				float dx = mpx - p[idxj].x; dx = sep_cuda_wrap(dx, lbox.x);
				float dy = mpy - p[idxj].y; dy = sep_cuda_wrap(dy, lbox.y);
				float dz = mpz - p[idxj].z; dz = sep_cuda_wrap(dz, lbox.z);
				
				
				float distSqr = dx*dx + dy*dy + dz*dz;

				if ( distSqr < 2.0*FLT_EPSILON ) continue; // Self contribution
				
				if ( distSqr < cfsqr ) {
						
					if ( shift < nneighmax )
						neighlist[arrayOffset + shift] = idxj;
					else if ( shift >= nneighmax ) {
						printf("Neighbour list generation failed\n");
						return;
					}	
					
					shift++;
				}
			}

			__syncthreads();
			
		}
	}
}
	
/* Neighbourlist for particles excluding particles in same molecule */
__global__ void sep_cuda_build_neighblist(int *neighlist, float *dist, float4 *p, int *molindex, 
										  float cf, float3 lbox, unsigned nneighmax, unsigned npart) {

	int pidx = blockDim.x * blockIdx.x + threadIdx.x;
		
	if ( pidx < npart ){
		float cfsqr = cf*cf; 
		int arrayOffset = pidx*nneighmax;
		int moli = molindex[pidx];
		float mpx = __ldg(&p[pidx].x); float mpy = __ldg(&p[pidx].y); float mpz = __ldg(&p[pidx].z);

		#pragma unroll	
		for ( int n=0; n<nneighmax; n++ ) neighlist[arrayOffset + n] = -1; //<- this should be optimized 
		
		// Reset the distance traveled since last update
		dist[pidx] = 0.0f;
		
		int shift = 0;
		for ( int tile = 0; tile < gridDim.x; tile++ ) {

			for ( int j = 0; j < SEP_CUDA_NTHREADS; j++ ) {
				int idxj = tile*blockDim.x + j;
				
				if ( idxj >= npart )  break;
				
				if ( moli == molindex[idxj] ) continue;
				
				float dx = mpx - p[idxj].x; dx = sep_cuda_wrap(dx, lbox.x);
				float dy = mpy - p[idxj].y; dy = sep_cuda_wrap(dy, lbox.y);
				float dz = mpz - p[idxj].z; dz = sep_cuda_wrap(dz, lbox.z);
				
				float distSqr = dx*dx + dy*dy + dz*dz;

				if ( distSqr < 2.0*FLT_EPSILON ) continue; // Self contribution
				
				if ( distSqr < cfsqr ) {
						
					if ( shift < nneighmax )
						neighlist[arrayOffset + shift] = idxj;
					else if ( shift >= nneighmax ) {
						printf("Neighbour list generation failed\n");
						return;
					}	
					
					shift++;
				}
			}
			// __syncthreads();
		}
	}
}

/* Pair interactions - types specified */
__global__ void sep_cuda_lj(const char type1, const char type2, float3 params, int *neighblist, float4 *pos, float4 *force,
							int *molindex, float *epot, float4 *press, unsigned maxneighb, float3 lbox, const unsigned npart){

	
	int pidx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if ( pidx < npart ) {
		
		int itype = __float2int_rd(force[pidx].w);
		int atype = (int)type1; int btype = (int)type2; //cast is stupid!
		//int midx = molindex[pidx];
		
		if ( itype != atype && itype != btype ) return;
		
		float sigma = params.x; 
		float epsilon = params.y; 
		float cf = params.z; //__ldg does not work..?
		float cfsqr = cf*cf;
		float Epot_shift = 4.0*epsilon*(powf(sigma/cf, 12.) - powf(sigma/cf,6.));
		
		int offset = pidx*maxneighb;
			
		float mpx = __ldg(&pos[pidx].x); float mpy = __ldg(&pos[pidx].y); float mpz = __ldg(&pos[pidx].z);
				
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f; 
		float Epot = 0.0f; 
		float4 mpress; mpress.x = mpress.y = mpress.z = mpress.w = 0.0f;

		int n = 0;
		while ( neighblist[n+offset] != -1 ){
			int pjdx = neighblist[n+offset];
			int jtype = __float2int_rd(force[pjdx].w);
			//int mjdx = molindex[pjdx];
			
			if ( (itype == atype && jtype == btype) || (itype == btype && jtype == atype) ){
				
				float dx = mpx - pos[pjdx].x; dx = sep_cuda_wrap(dx, lbox.x);
				float dy = mpy - pos[pjdx].y; dy = sep_cuda_wrap(dy, lbox.y);
				float dz = mpz - pos[pjdx].z; dz = sep_cuda_wrap(dz, lbox.z);

				float distSqr = dx*dx + dy*dy + dz*dz;

				if ( distSqr < cfsqr ) {
					float rri = sigma*sigma/distSqr; 
					float rri3 = rri*rri*rri;
					float ft = 48.0*epsilon*rri3*(rri3 - 0.5)*rri;
				
					Fx += ft*dx; Fy += ft*dy; Fz += ft*dz;
					Epot += 0.5*(4.0*epsilon*rri3*(rri3 - 1.0) - Epot_shift);
					
					// pidx not in molecule (atom. press)
					//if ( midx == - 1 ){ 
						mpress.x += dx*ft*dx + dy*ft*dy + dz*ft*dz; 
						mpress.y += dx*ft*dy; mpress.z += dx*ft*dz; mpress.w += dy*ft*dz;
					//}
					// else pidx/pjdx not in same molecule (mol. press)
					//else if ( midx != mjdx ){
				//		mpress.x += ft*dx; mpress.y += ft*dy; mpress.z += ft*dz; 
				//	}
					
				}
			}
			
			n++;
		}
		
		force[pidx].x += Fx; force[pidx].y += Fy; force[pidx].z += Fz; 
		epot[pidx] += Epot; 
		
		//if ( midx == -1 ){
			press[pidx].x += mpress.x;
			press[pidx].y += mpress.y; press[pidx].z += mpress.z; press[pidx].w += mpress.w; 
		/*}
		else {
			atomicAdd(&(press[midx].x), mpress.x); 
			atomicAdd(&(press[midx].y), mpress.y);
			atomicAdd(&(press[midx].z), mpress.z);
		}*/

	}
		
}


/* Pair interactions - all types have same interactions (faster) */
__global__ void sep_cuda_lj(float3 params, int *neighblist, float4 *pos, float4 *force,
							float *epot, float4 *press, unsigned maxneighb, float3 lbox, const unsigned npart){

	
	int pidx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if ( pidx < npart ) {
		
		float sigma = params.x; 
		float epsilon = params.y; 
		float cf = params.z; //__ldg does not work..?
		float cfsqr = cf*cf;
		float Epot_shift = 4.0*epsilon*(powf(sigma/cf, 12.) - powf(sigma/cf,6.));
		
		int offset = pidx*maxneighb;
			
		float mpx = __ldg(&pos[pidx].x); float mpy = __ldg(&pos[pidx].y); float mpz = __ldg(&pos[pidx].z);
				
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f; 
		float Epot = 0.0f; 
		float4 mpress; mpress.x = mpress.y = mpress.z = mpress.w = 0.0f;
		int n = 0;
		while ( neighblist[n+offset] != -1 ){
			int pjdx = neighblist[n+offset];
				
			float dx = mpx - pos[pjdx].x; dx = sep_cuda_wrap(dx, lbox.x);
			float dy = mpy - pos[pjdx].y; dy = sep_cuda_wrap(dy, lbox.y);
			float dz = mpz - pos[pjdx].z; dz = sep_cuda_wrap(dz, lbox.z);

			float distSqr = dx*dx + dy*dy + dz*dz;

			if ( distSqr < cfsqr ) {
				float rri = sigma*sigma/distSqr; 
				float rri3 = rri*rri*rri;
				float ft =  48.0*epsilon*rri3*(rri3 - 0.5)*rri; //pow( sqrtf(1.0/distSqr), 11.0 ); //
				
				Fx += ft*dx; Fy += ft*dy; Fz += ft*dz;
				Epot += 0.5*(4.0*epsilon*rri3*(rri3 - 1.0) - Epot_shift);
				mpress.x += dx*ft*dx + dy*ft*dy + dz*ft*dz; 
				mpress.y += dx*ft*dy; mpress.z += dx*ft*dz; mpress.w += dy*ft*dz;
			}
			
			n++;
		}
			
		
			
		force[pidx].x += Fx; force[pidx].y += Fy; force[pidx].z += Fz;
		epot[pidx] += Epot; 
		press[pidx].x += mpress.x;
		press[pidx].y += mpress.y; press[pidx].z += mpress.z; press[pidx].w += mpress.w; 
	}
}



__global__ void sep_cuda_lj_sf(const char type1, const char type2, float3 params, int *neighblist, float4 *pos, float4 *force,
								float *epot, float4 *press, unsigned maxneighb, float3 lbox, const unsigned npart){

	
	int pidx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if ( pidx < npart ) {
		
		int itype = __float2int_rd(force[pidx].w);
		int atype = (int)type1; int btype = (int)type2; //cast stupid
		
		if ( itype != atype && itype != btype ) return;
		
		float sigma = params.x; 
		float epsilon = params.y; 
		float cf = params.z; //__ldg does not work..?
		float cfsqr = cf*cf; 
		float Epot_shift = 4.0*epsilon*(powf(sigma/cf, 12.) - powf(sigma/cf,6.));
		float force_shift = 48.0*epsilon*powf(sigma/cf,6.0)*(powf(sigma/cf,3.0) - 0.5)*pow(sigma/cf, 2.0);
		
		int offset = pidx*maxneighb;
			
		float mpx = __ldg(&pos[pidx].x); float mpy = __ldg(&pos[pidx].y); float mpz = __ldg(&pos[pidx].z);
				
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f; 
		float Epot = 0.0f; 
		float4 mpress; mpress.x = mpress.y = mpress.z = mpress.w = 0.0f;
		int n = 0;
		while ( neighblist[n+offset] != -1 ){
			int pjdx = neighblist[n+offset];
			int jtype = __float2int_rd(force[pjdx].w);
			
			if ( (itype == atype && jtype == btype) || (itype == btype && jtype == atype) ){
				
				float dx = mpx - pos[pjdx].x; dx = sep_cuda_wrap(dx, lbox.x);
				float dy = mpy - pos[pjdx].y; dy = sep_cuda_wrap(dy, lbox.y);
				float dz = mpz - pos[pjdx].z; dz = sep_cuda_wrap(dz, lbox.z);

				float distSqr = dx*dx + dy*dy + dz*dz;

				if ( distSqr < cfsqr ) {
					float rri = sigma*sigma/distSqr; 
					float rri3 = rri*rri*rri;
					float ft = 48.0*epsilon*rri3*(rri3 - 0.5)*rri + force_shift;
				
					Fx += ft*dx; Fy += ft*dy; Fz += ft*dz;
					Epot += 0.5*(4.0*epsilon*rri3*(rri3 - 1.0) - Epot_shift);
					mpress.x += dx*ft*dx + dy*ft*dy + dz*ft*dz; 
					mpress.y += dx*ft*dy; mpress.z += dx*ft*dz; mpress.w += dy*ft*dz;
				}
			}
			
			n++;
		}
		
		force[pidx].x += Fx; force[pidx].y += Fy; force[pidx].z += Fz;
		epot[pidx] += Epot; 
		press[pidx].x += mpress.x;
		press[pidx].y += mpress.y; press[pidx].z += mpress.z; press[pidx].w += mpress.w; 
	}
		
}



__global__ void sep_cuda_sf(float cf, int *neighblist, float4 *pos, float4 *vel, float4 *force,
							float *epot, float4 *press, unsigned maxneighb, float3 lbox, const unsigned npart){
	
	__const__ int pidx = blockDim.x * blockIdx.x + threadIdx.x;
	
	if ( pidx < npart ) {
		
		float cfsqr = cf*cf;
		float icf = 1.0/cf;
		float icf2 = 1.0/cfsqr;
		
		int offset = pidx*maxneighb;
			
		float mpx = __ldg(&pos[pidx].x); 
		float mpy = __ldg(&pos[pidx].y); 
		float mpz = __ldg(&pos[pidx].z);
				
		float Fx = 0.0; float Fy = 0.0; float Fz = 0.0; float Epot = 0.0;		
		float4 mpress; mpress.x = mpress.y = mpress.z = mpress.w = 0.0f;
		
		int n = 0;
		while ( neighblist[n+offset] != -1 ){
			int pjdx = neighblist[n+offset];
				
			float dx = mpx - pos[pjdx].x; dx = sep_cuda_wrap(dx, lbox.x);
			float dy = mpy - pos[pjdx].y; dy = sep_cuda_wrap(dy, lbox.y);
			float dz = mpz - pos[pjdx].z; dz = sep_cuda_wrap(dz, lbox.z);

			float distSqr = dx*dx + dy*dy + dz*dz;

			if ( distSqr < cfsqr ) {
				float zizj = vel[pidx].w*vel[pjdx].w;
				float dist = sqrtf(distSqr); 
				float ft = zizj*(1.0/distSqr - icf2)/dist; 
				
				Fx += ft*dx; Fy += ft*dy; Fz += ft*dz;
				
				Epot += 0.5*zizj*(1.0/dist + (dist-cf)*icf2 - icf);
				mpress.x += dx*ft*dx + dy*ft*dy + dz*ft*dz; 
				mpress.y += dx*ft*dy; mpress.z += dx*ft*dz; mpress.w += dy*ft*dz;
			}

			n++;
		}
		
		force[pidx].x += Fx; force[pidx].y += Fy; force[pidx].z += Fz;
		epot[pidx] += Epot;	
		press[pidx].x += mpress.x;
		press[pidx].y += mpress.y; press[pidx].z += mpress.z; press[pidx].w += mpress.w;
	}	
		
}


void sep_cuda_force_lj(sepcupart *pptr, const char types[], float params[3]){
	const int nb = pptr->nblocks; 
	const int nt = pptr->nthreads;
	
	float3 ljparams = make_float3(params[0],params[1],params[2]);
	
	sep_cuda_lj<<<nb, nt>>>
		(types[0], types[1], ljparams, pptr->neighblist, pptr->dx, pptr->df, pptr->dmolindex, 
					pptr->epot, pptr->press, pptr->maxneighb, pptr->lbox, pptr->npart);
	
	hipDeviceSynchronize();

}

void sep_cuda_force_lj(sepcupart *pptr, float params[3]){
	const int nb = pptr->nblocks; 
	const int nt = pptr->nthreads;
	
	float3 ljparams = make_float3(params[0],params[1],params[2]);
	
	sep_cuda_lj<<<nb, nt>>>
		(ljparams, pptr->neighblist, pptr->dx, pptr->df, pptr->epot, pptr->press, pptr->maxneighb, pptr->lbox, pptr->npart);
		
	hipDeviceSynchronize();

}


void sep_cuda_force_lj_sf(sepcupart *pptr, const char types[], float params[3]){
	const int nb = pptr->nblocks; 
	const int nt = pptr->nthreads;
	
	float3 ljparams = make_float3(params[0],params[1],params[2]);
	
	sep_cuda_lj_sf<<<nb, nt>>>
		(types[0], types[1], ljparams, pptr->neighblist, pptr->dx, pptr->df, pptr->epot, 
											pptr->press, pptr->maxneighb, pptr->lbox, pptr->npart);
	hipDeviceSynchronize();

}



void sep_cuda_force_sf(sepcupart *pptr, const float cf){
	const int nb = pptr->nblocks; 
	const int nt = pptr->nthreads;
	
	sep_cuda_sf<<<nb,nt>>>
		(cf, pptr->neighblist, pptr->dx, pptr->dv, pptr->df, pptr->epot, 
											pptr->press, pptr->maxneighb, pptr->lbox, pptr->npart);
	hipDeviceSynchronize();

}

void sep_cuda_update_neighblist(sepcupart *pptr, sepcusys *sptr, float maxcutoff){
	const int nb = sptr->nblocks; 
	const int nt = sptr->nthreads;

	if ( pptr->hexclusion_rule == SEP_CUDA_EXCL_NONE ) {
		sep_cuda_build_neighblist<<<nb, nt>>>
			(pptr->neighblist, pptr->dx, pptr->ddist, sptr->skin+maxcutoff, pptr->lbox, pptr->maxneighb,pptr->npart);
	}
	else if ( pptr->hexclusion_rule == SEP_CUDA_EXCL_MOLECULE ) {
		sep_cuda_build_neighblist<<<nb, nt>>>
			(pptr->neighblist, pptr->ddist, pptr->dx, pptr->dmolindex, sptr->skin+maxcutoff, pptr->lbox, pptr->maxneighb,pptr->npart);
	}
	else {
		fprintf(stderr, "Exclusion rule invalid");
	}
		
	hipDeviceSynchronize();

}


