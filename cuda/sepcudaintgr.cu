#include "hip/hip_runtime.h"

#include "sepcudaintgr.h"


#ifdef OCTAVE
__inline__ __device__ float sep_cuda_dot(float4 a){
	
	return (a.x*a.x + a.y*a.y + a.z*a.z);
	
}

__global__ void sep_cuda_sumenergies(float3 *totalsum, float4* dx, float4 *dv, float4 *df, 
									 float dt, float *epot, unsigned npart){

	int id = blockIdx.x*blockDim.x + threadIdx.x;
	__shared__ float3 sums;
	
	if ( threadIdx.x==0 ) {
		sums.x = sums.y = sums.z = 0.0f;
	}
	__syncthreads();

	if ( id < npart ){
		float4 vel; 
		vel.x =  dv[id].x - 0.5*dt*df[id].x/dx[id].w;
		vel.y =  dv[id].y - 0.5*dt*df[id].y/dx[id].w;
		vel.z =  dv[id].z - 0.5*dt*df[id].z/dx[id].w;
		
		float mykin = 0.5*sep_cuda_dot(vel)*dx[id].w;
		float mymom = (dv[id].x + dv[id].y + dv[id].z)*dx[id].w;
		
		atomicAdd(&sums.x, mykin);
		atomicAdd(&sums.y, epot[id]);
		atomicAdd(&sums.z, mymom);
	}

	__syncthreads();
	
	if ( id < npart && threadIdx.x == 0 ) {
		atomicAdd(&(totalsum->x), sums.x);
		atomicAdd(&(totalsum->y), sums.y);
		atomicAdd(&(totalsum->z), sums.z);
	}
	
}

#endif

__inline__ __device__ float sep_cuda_wrap(float x, float lbox){
	
	if ( x > 0.5*lbox ) 
		x -= lbox;
	else if  ( x < -0.5*lbox ) 
		x += lbox;
	
	return x;
}

__inline__ __device__ float sep_cuda_periodic(float x, float lbox, int *crossing){
	
	if ( x > lbox ) {
		x -= lbox;  
		*crossing = *crossing + 1;
	}
	else if  ( x < 0 ) {
		x += lbox;
		*crossing = *crossing - 1;
	}
	
	return x;
}



__global__ void sep_cuda_leapfrog(float4 *pos, float4 *vel, 
		  float4 *force, float *dist, int3 *crossing, float dt, float3 lbox, unsigned npart){

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	float4 oldpos = make_float4(pos[i].x, pos[i].y, pos[i].z, 0.0f);
	float4 mypos = make_float4(pos[i].x, pos[i].y, pos[i].z, pos[i].w);

	if ( i < npart ) {
		float imass = 1.0/mypos.w;
		
		vel[i].x += force[i].x*imass*dt; 
		vel[i].y += force[i].y*imass*dt;
		vel[i].z += force[i].z*imass*dt;
		
		mypos.x += vel[i].x*dt;
		mypos.x = sep_cuda_periodic(mypos.x, lbox.x, &(crossing[i].x));
		
		mypos.y += vel[i].y*dt;
		mypos.y = sep_cuda_periodic(mypos.y, lbox.y, &(crossing[i].y));
	
		mypos.z += vel[i].z*dt;
		mypos.z = sep_cuda_periodic(mypos.z, lbox.z, &(crossing[i].z));
					
		float dx = oldpos.x - mypos.x; dx = sep_cuda_wrap(dx, lbox.x);
		float dy = oldpos.y - mypos.y; dy = sep_cuda_wrap(dy, lbox.y);
		float dz = oldpos.z - mypos.z; dz = sep_cuda_wrap(dz, lbox.z);
	
		dist[i] += sqrtf(dx*dx + dy*dy + dz*dz);

		pos[i].x = mypos.x; pos[i].y = mypos.y; pos[i].z = mypos.z;
	}
	
}


__global__ void sep_cuda_update_nosehoover(float *alpha, float3 *denergies, float temp0, 
										   float tau, float dt, unsigned int npart){

	float temp = (2.0/3.0)*denergies->x/npart; 

	*alpha = *alpha + dt/(tau*tau)*(temp/temp0 - 1.0);

}


__global__ void sep_cuda_nosehoover(float *alpha, float4 *pos, float4 *vel, float4 *force, unsigned npart){
	
	unsigned id = blockIdx.x*blockDim.x + threadIdx.x;

	if ( id < npart ){	
		float fac = (*alpha)*pos[id].w;
		force[id].x -= fac*vel[id].x; 
		force[id].y -= fac*vel[id].y; 
		force[id].z -= fac*vel[id].z;		
	}
}

void sep_cuda_thermostat_nh(sepcupart *pptr, sepcusys *sptr, float temp0, float tau){
	const int nb = sptr->nblocks; 
	const int nt = sptr->nthreads;
	
	// Get current system kinetic energy
	sep_cuda_sumenergies<<<nb,nt>>>
		(sptr->denergies, pptr->dx, pptr->dv, pptr->df, sptr->dt, pptr->epot, sptr->npart);
	hipDeviceSynchronize();
	
	// Update nh-alpha dynamics (single thread)
	sep_cuda_update_nosehoover<<<1,1>>>
		(sptr->dalpha, sptr->denergies, temp0, tau, sptr->dt, sptr->npart);
	hipDeviceSynchronize();

	// Add thermostat force
	sep_cuda_nosehoover<<<nb, nt>>>
		(sptr->dalpha, pptr->dx, pptr->dv, pptr->df, sptr->npart);
	hipDeviceSynchronize();		
	
}

   
void sep_cuda_integrate_leapfrog(sepcupart *pptr, sepcusys *sptr){
	const int nb = sptr->nblocks; 
	const int nt = sptr->nthreads;

	sep_cuda_leapfrog<<<nb, nt>>>
		(pptr->dx, pptr->dv, pptr->df, pptr->ddist, pptr->dcrossings, sptr->dt, pptr->lbox, pptr->npart);
	hipDeviceSynchronize();
	
}


